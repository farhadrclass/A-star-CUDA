#include "hip/hip_runtime.h"
#include <stdio.h>

#include "astar_gpu.h"
#include "heap.h"
#include "list.h"
#include "sliding_puzzle.h"
#include "cuda_utils.h"

#define STATES (1024 * 1024)
#define HASH_SIZE  (1024 * 1024)
#define HASH_FUNS 128

__global__ void astar_kernel(const char *s, const char *t, int k,
		heap **Q, list *S, state **H, state *states_pool, char **expand_buf,
		expand_fun expand, heur_fun h);
__device__ void hash_with_replacement_deduplicate(state **H, list *T);
__device__ int f(const state *x, const char *t, heur_fun h);


char **expand_buf_create(int elements, int element_size);

state *states_pool_create();
void states_pool_destroy(state *states_pull);
__device__ state *state_create(const char *node, int f, int g, state *prev,
		state *states_pool);

__device__ void print_expanded(char **expanded) {
	for (int i = 0; expanded[i] != NULL; i++) {
		printf("%s\n", expanded[i]);
	}
}


int astar_gpu(const char *s_in, const char *t_in, int k) {
	char *s_gpu, *t_gpu;
	expand_fun expand_fun_cpu;
	heur_fun h_cpu;
	int expand_elements;
	int expand_element_size;

	sliding_puzzle_preprocessing(s_in, t_in, &s_gpu, &t_gpu, &expand_fun_cpu, &h_cpu,
			&expand_elements, &expand_element_size);

	state **H;
	char **expand_buf = expand_buf_create(expand_elements, expand_element_size);
	HANDLE_RESULT(hipMalloc(&H, HASH_SIZE * sizeof(state*)));
	heap **Q = heaps_create(k);
	list *S = list_create(100);
	state *states_pool = states_pool_create();

	astar_kernel<<<1, 1>>>(s_gpu, t_gpu, k, Q, S, H, states_pool, expand_buf,
			expand_fun_cpu, h_cpu);

	states_pool_destroy(states_pool);
	list_destroy(S);
	heaps_destroy(Q, k);
	HANDLE_RESULT(hipFree(H));
	HANDLE_RESULT(hipDeviceSynchronize());
	return 0;
}

__global__ void astar_kernel(const char *s, const char *t, int k,
		heap **Q, list *S, state **H, state *states_pool, char **expand_buf,
		expand_fun expand, heur_fun h) {
	state *m = NULL;

	heap_insert(Q[0], state_create(s, 0, 0, NULL, states_pool));

	int steps = 0;
	while (!heaps_empty(Q, k)) {
		steps++;
		list_clear(S);
		for (int i = 0; i < k; i++) {
			if (Q[i]->size == 0) continue;
			state *q = heap_extract(Q[i]);
			//printf("%s\n", q->node);
			if (steps % 1000 == 0) printf("distance: %d\n", h(q->node, t));
			if (cuda_str_eq(q->node, t)) {
				if (m == NULL || f(q, t, h) < f(m, t, h)) {
					m = q;
				}
				continue;
			}
			expand(q->node, expand_buf);
			for (int j = 0; expand_buf[j] != NULL; j++) {
				list_insert(S, state_create(expand_buf[j], -1, q->g + 1, q, states_pool));
			}
		}
		if (m != NULL && f(m, t, h) < heaps_min(Q, k)) {
			printf("In %d steps: Found path of length %d: [\n", steps, m->g);
			state *cur = m;
			while (cur != NULL) {
				for (int i = 0; i < 25; i++) {
					printf("%c%c ", cur->node[3 * i], cur->node[3 * i + 1]);
					if (i % 5 == 4) printf("\n");
				}
				printf("\n");
				cur = cur->prev;
			}
			printf("]\n");
			break;
		}
		hash_with_replacement_deduplicate(H, S);
		for (int i = 0; i < S->length; i++) {
			state *t1 = list_get(S, i);
			if (t1 != NULL) {
				t1->f = f(t1, t, h);
				heap_insert(Q[0], t1);
			}
		}
	}
}

__device__ void hash_with_replacement_deduplicate(state **H, list *T) {
	for (int i = 0; i < T->length; i++) {
		int z = 0;
		state *t = list_get(T, i);
		for (int j = 0; j < HASH_FUNS; j++) {
			state *el = H[jenkins_hash(j, t->node) % HASH_SIZE];
			if (el == NULL || cuda_str_eq(t->node, el->node)) {
				z = j;
				break;
			}
		}
		int index = jenkins_hash(z, t->node) % HASH_SIZE;
		t = (state*)atomicExch((unsigned long long*)&(H[index]), (unsigned long long)t);
		if (t != NULL && cuda_str_eq(t->node, list_get(T, i)->node)) {
			list_remove(T, i);
			continue;
		}
		t = list_get(T, i);
		for (int j = 0; j < HASH_FUNS; j++) {
			if (j != z) {
				state *el = H[jenkins_hash(j, t->node) % HASH_SIZE];
				if (el != NULL && cuda_str_eq(el->node, t->node)) {
					list_remove(T, i);
					break;
				}
			}
		}
	}
}

__device__ int f(const state *x, const char *t, heur_fun h) {
	return x->g + h(x->node, t);
}

state *states_pool_create() {
	state *states_pool = NULL;
	HANDLE_RESULT(hipMalloc(&states_pool, STATES * sizeof(state)));
	HANDLE_RESULT(hipMemset(states_pool, 0, STATES * sizeof(state)));
	return states_pool;
}

void states_pool_destroy(state *states_pull) {
	HANDLE_RESULT(hipFree(states_pull));
}

char **expand_buf_create(int elements, int element_size) {
	elements++; // For terminating NULL element
	element_size++; // For terminating NULL char
	char **buf_cpu = (char**)malloc(elements * sizeof(char*));
	for (int i = 0; i < elements; i++) {
		HANDLE_RESULT(hipMalloc(&(buf_cpu[i]), element_size));
	}
	char **buf_gpu;
	HANDLE_RESULT(hipMalloc(&buf_gpu, elements * sizeof(char*)));
	HANDLE_RESULT(hipMemcpy(buf_gpu, buf_cpu, elements * sizeof(char),
				hipMemcpyDefault));
	HANDLE_RESULT(hipDeviceSynchronize());
	free(buf_cpu);
	return buf_gpu;

}

__device__ int used_states = 0;
__device__ state *state_create(const char *node, int f, int g, state *prev,
		state *states_pool) {
	int index = atomicAdd(&used_states, 1);
	state *result = &(states_pool[index]);
	result->node = node;
	result->f = f;
	result->g = g;
	result->prev = prev;
	return result;
}

